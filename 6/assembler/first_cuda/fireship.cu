
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
Terms:
Block: A group of threads
Cuda kernel: code that runs on the gpu

Syntax:
__global__ a function type that indicates the function is a cuda kernel
threadIdx: Varibale avialble in cuda kernels. Each thread has its index
blockDim: Variable avialable in cuda kernels.
	
*/
 
// __global__ indicates that this function is run on gpu
__global__ void add(int *a, int *b, int *c)
{
	//threadIdx = buildin varaible that changes on runtime
	//is the threads id that is executing the job (so many of the same
	// threadIdx exist at the same time)
	//blockDim = 
	int i = threadIdx.x + blockDim.x;
	c[i] = a[i] + b[i];
}

// __mangaged__ vars can be accessed by cpu and gpu
__managed__ int vector_a[256], vector_b[256], vector_c[256];


void	test(void)
{
	for (int i = 0; i < 256; i++)
	{
		vector_a[i] = i;
		vector_b[i] = 256 - i;
	}
	//<< >>: configures the 'block' count and the threads per block
	add<<<1, 256>>>(vector_a, vector_b, vector_c);
	hipDeviceSynchronize();
}

int main(void)
{

	return (0);
}

