#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fcntl.h>
#include <unistd.h>
#include <errno.h>
#include <string.h>

#define BUFF_SIZE 100000

__global__ void remove_spaces_gpu(char *buffer, int read_bytes, int *byte_count)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < read_bytes) {
        if (buffer[idx] != ' ') {
            int pos = atomicAdd(byte_count, 1);
            buffer[pos] = buffer[idx];
        }
    }
}

int main() {
    char hostBuffer[BUFF_SIZE];     // Original buffer on host
    char *devBuffer;      // Buffer on device (GPU)
    int read_bytes = 0; // Number of bytes to process
    int byte_count = 0;
    int *devByteCount;    // To store the result on device

	int fd = open("test_cuda.cu", O_RDONLY);
	if (errno)
		printf("%s\n", strerror(errno));
	errno = 0;
	read_bytes = read(fd ,hostBuffer,BUFF_SIZE);
	if (errno)
		printf("%s\n", strerror(errno));
	errno = 0;
	hostBuffer[read_bytes++] = 0;
	printf("read bytes: %d\n", read_bytes);
	printf("base: %s\n", hostBuffer);
    hipMalloc((void **)&devBuffer, read_bytes);
    hipMalloc((void **)&devByteCount, sizeof(int));

    hipMemcpy(devBuffer, hostBuffer, read_bytes, hipMemcpyHostToDevice);
    hipMemcpy(devByteCount, &byte_count, sizeof(int), hipMemcpyHostToDevice);

    // Kernel launch parameters: Adjust these values based on your needs
    int threadsPerBlock = 256;
    int blocks = (read_bytes + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    remove_spaces_gpu<<<blocks, threadsPerBlock>>>(devBuffer, read_bytes, devByteCount);

    // Copy back the results
    hipMemcpy(hostBuffer, devBuffer, read_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(&byte_count, devByteCount, sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(devBuffer);
    hipFree(devByteCount);
	printf("================================================================================================================\n");
	printf("count after: %d\n", byte_count);
	printf("result: %s\n", hostBuffer);
    return 0;
}
